
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_from_gpu()
{
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    printf("hello world from the gpu from block id %d, thread id %d \n", bid, tid);
}

int main() 
{
    hello_from_gpu<<<33,5>>>();
    hipDeviceSynchronize();
    return 0;
}